#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>
#include <hipblas.h>

hipblasHandle_t handle;

void createCublas() {
    hipblasCreate(&handle);
}

void destroyCublas() {
    hipblasDestroy(handle);
}

void matmul_cublas(float* result, float* a, float* b, int m, int n, int k) {
    // Calculate with Cublas
    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,
        &alpha,
        b, n,
        a, k,
        &beta,
        result, n);
}